#include "hip/hip_runtime.h"
#include <assert.h>
#include "ht_op.h"
#include "ht_flag.h"

int HT_stream_op_mode = HT_MODE_KERNEL;

__global__ void HT_kernel_set(volatile uint64_t* var, uint64_t val)
{
    *var = val;
}

__global__ void HT_kernel_wait(volatile uint64_t* var, uint64_t val)
{
    while(*var != val);
}

typedef struct {
    HT_flag_t *flag;
    uint64_t val;
} host_fn_params;

void HT_host_fn_set(host_fn_params *params)
{
    params->flag->host_val = params->val;
    free(params);
}

void HT_host_fn_wait(host_fn_params *params)
{
    while(params->flag->host_val != params->val);
    free(params);
}

void HT_set(HT_flag_t* flag, uint64_t val, hipStream_t stream)
{
    switch (HT_stream_op_mode) {
        case HT_MODE_HOST_FN:
            {
                host_fn_params *params = (host_fn_params *) malloc(sizeof(host_fn_params));
                params->flag = flag;
                params->val = val;
                hipLaunchHostFunc(stream, (hipHostFn_t) HT_host_fn_set, params);
            }
            break;
        case HT_MODE_KERNEL:
            HT_kernel_set<<<1,1,0,stream>>>(flag->dev_ptr, val);
            break;
        case HT_MODE_STREAM_MEM_OP:
            hipStreamWriteValue64(stream, (hipDeviceptr_t) flag->dev_ptr, val, 0);
            break;
        default:
            assert(0);
    }
}

void HT_wait(HT_flag_t* flag, uint64_t val, hipStream_t stream)
{
    switch (HT_stream_op_mode) {
        case HT_MODE_HOST_FN:
            {
                host_fn_params *params = (host_fn_params *) malloc(sizeof(host_fn_params));
                params->flag = flag;
                params->val = val;
                hipLaunchHostFunc(stream, (hipHostFn_t) HT_host_fn_wait, params);
            }
            break;
        case HT_MODE_KERNEL:
            HT_kernel_wait<<<1,1,0,stream>>>(flag->dev_ptr, val);
            break;
        case HT_MODE_STREAM_MEM_OP:
            hipStreamWaitValue64(stream, (hipDeviceptr_t) flag->dev_ptr, val, hipStreamWaitValueEq);
            break;
        default:
            assert(0);
    }
}
